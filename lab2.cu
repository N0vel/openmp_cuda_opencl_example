#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "debice_launch_parameters"
#include <iostream>
#include "BMP.h"
#include <chrono>

using namespace std;

#define BSIZE 32
#define input_image_size 0

__global__  to_grayscale(BMP filtered_image)
{
    int low = filtered_image.bmp_info_header.height*threadnum/numthreads, high = filtered_image.bmp_info_header.height*(threadnum+1)/numthreads;
    double gray_value = 0.;
    uint32_t channels = filtered_image.bmp_info_header.bit_count / 8;
    
    return filtered_image
}





int main() 
{
	using clock = std::chrono::system_clock;
	using ms = std::chrono::milliseconds;
    char images [3][50] = {"1024x768.bmp", "1280x960.bmp", "2048x1536.bmp"};
    bitmap_image bmp(images[input_image_size]);
    unsigned int h = bmp.height()
    unsigned int w = bmp.width()

    const auto before = clock::now();

    const auto duration = std::chrono::duration_cast<ms>(clock::now() - before);
    cout << "Time passed: " << duration / 1000. << " ms"
	return 0;
}