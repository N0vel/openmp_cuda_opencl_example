#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include "bitmap_image.hpp"
#include <ctime>
#include <stdio.h>
using namespace std;

#define BSIZE 16
#define input_image_size 2

void cudaCheckErrors(string msg)
{
	while (0)
	{
		hipError_t __err = hipGetLastError();
		if (__err != hipSuccess)
		{
		fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);
		fprintf(stderr, "*** FAILED - ABORTING\n");
		system("pause");
		exit(1);
		}
	}
}

void check_cuda_devices()
{
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
		prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
		prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

}

__global__ void gauss_filter(float *input, float *output, int columns, int rows)
{
	int row = threadIdx.y + blockDim.y*blockIdx.y;
	int col = threadIdx.x + blockDim.x*blockIdx.x;
	if ((row > 0) && (row < rows - 1) && (col > 0) && (col < columns-1))
	{
		output[(row-1)  * (columns-2) + (col-1)] = 0.0;
		output[(row-1)  * (columns-2) + (col-1)] += 1./16. * input[(row-1) * columns + col - 1];
		output[(row-1)  * (columns-2) + (col-1)] += 2./16. * input[(row-1) * columns + col];
		output[(row-1)  * (columns-2) + (col-1)] += 1./16. * input[(row-1) * columns + col + 1];
		output[(row-1)  * (columns-2) + (col-1)] += 2./16. * input[row * columns + col - 1];
		output[(row-1)  * (columns-2) + (col-1)] += 4./16. * input[row * columns + col];
		output[(row-1)  * (columns-2) + (col-1)] += 2./16. * input[row * columns + col + 1];
		output[(row-1)  * (columns-2) + (col-1)] += 1./16. * input[(row+1) * columns + col - 1];
		output[(row-1)  * (columns-2) + (col-1)] += 2./16. * input[(row+1) * columns + col];
		output[(row-1)  * (columns-2) + (col-1)] += 1./16. * input[(row+1) * columns + col + 1];
	}
}

float* rgb_to_gray(bitmap_image image)
{
	int h = image.height();
	int w = image.width();
    unsigned char r, g, b;
	float *gray = new float[w*h];
	for (int y = 0; y < h; y++)
	{
		for (int x = 0; x < w; x++)
		{
			image.get_pixel(x, y, r, g, b);
			gray[y*w+x] = (float) ((r+g+b) / 3.);
		}

	}
	return gray;
}


int main() 
{
	check_cuda_devices();
	cout << "BLOCK SIZE: "  << BSIZE << endl;
	cout << "Image index: " << input_image_size << endl;
    char images [3][50] = {"1024x768.bmp", "1280x960.bmp", "2048x1536.bmp"};
    bitmap_image image(images[input_image_size]);
    int h = image.height();
    int w = image.width();
    // RGB to gray
    float *gray = rgb_to_gray(image);
	clock_t start, stop;
	start = clock();
	// No padding: filtered image is smaller than source image
	float *gpu_gray, *gpu_filtered;
	hipMalloc((void**)&gpu_gray, sizeof(float)*(w*h));
	hipMalloc((void**)&gpu_filtered, sizeof(float)*((w-2)*(h-2)));
	hipMemcpy(gpu_gray, gray, sizeof(float)*(w*h), hipMemcpyKind::hipMemcpyHostToDevice);
	cudaCheckErrors("Err 1");
//	start = clock(); // computation only clock
	dim3 block(BSIZE, BSIZE);
	dim3 grid((w + block.x ) / block.x, (h + block.y) / block.y);

	gauss_filter <<< grid, block >>>(gpu_gray, gpu_filtered, w, h);

	hipDeviceSynchronize();
	cudaCheckErrors("Err 2");
//	stop = clock(); // computation only clock

	float *filtered = new float[(w - 2)*(h - 2)];
	hipMemcpy(filtered, gpu_filtered, sizeof(float)*(w - 2)*(h - 2),
	hipMemcpyKind::hipMemcpyDeviceToHost);
	hipFree(gpu_gray);
	hipFree(gpu_filtered);

	stop = clock();
    cout << "Time passed: " << (stop - start) / (double)CLOCKS_PER_SEC * 1000.0 << " ms";
    bitmap_image G(images[input_image_size]);
    int value;
    for (int y = 0; y < h - 2; y++)
	{
		for (int x = 0; x < w - 2; x++)
		{
			value = (int)(filtered[y*(w-2)+x]);
			G.set_pixel(x+1, y+1, value, value, value);
		}
	}
    G.save_image("output.bmp");
	return 0;
}
